#include "hip/hip_runtime.h"
﻿#ifndef _CUDA_CU
#define _CUDA_CU

#include "Header.cuh"
#include "hip/hip_runtime_api.h"
#include "";
#include "DXF.h"
#include "Voxel.h"

#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>

#include <chrono>
using std::chrono::duration_cast;
using std::chrono::microseconds;
typedef std::chrono::steady_clock the_clock;

//
//	Interpolate between two nodes
__device__ void AddInterpolatedVertex(unsigned int voxelIter,
	unsigned int vertexIter,
	float* devVertexArray,
	float* devNodePositionArray,
	float* devNodeDensityArray,
	unsigned int node1,
	unsigned int node2,
	float devDensityCutoff)
{
	if (abs(devDensityCutoff - devNodeDensityArray[(voxelIter * 8) + node1]) < 0.00001f)
	{
		devVertexArray[(vertexIter * 3)] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3)];
		devVertexArray[(vertexIter * 3) + 1] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 1];
		devVertexArray[(vertexIter * 3) + 2] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 2];
		return;
	}
	if (abs(devDensityCutoff - devNodeDensityArray[(voxelIter * 8) + node2]) < 0.00001f)
	{
		devVertexArray[(vertexIter * 3)] = devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3)];
		devVertexArray[(vertexIter * 3) + 1] = devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3) + 1];
		devVertexArray[(vertexIter * 3) + 2] = devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3) + 2];
		return;
	}
	if (abs(devNodeDensityArray[(voxelIter * 8) + node1] - devNodeDensityArray[(voxelIter * 8) + node2]) < 0.00001f)
	{
		devVertexArray[(vertexIter * 3)] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3)];
		devVertexArray[(vertexIter * 3) + 1] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 1];
		devVertexArray[(vertexIter * 3) + 2] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 2];
		return;
	}

	float mu = (devDensityCutoff - devNodeDensityArray[(voxelIter * 8) + node1]) / (devNodeDensityArray[(voxelIter * 8) + node2] - devNodeDensityArray[(voxelIter * 8) + node1]);
	
	devVertexArray[(vertexIter * 3)] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3)] + mu * (devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3)] - devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3)]);
	devVertexArray[(vertexIter * 3) + 1] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 1] + mu * (devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3) + 1] - devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 1]);
	devVertexArray[(vertexIter * 3) + 2] = devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 2] + mu * (devNodePositionArray[(voxelIter * (8 * 3)) + (node2 * 3) + 2] - devNodePositionArray[(voxelIter * (8 * 3)) + (node1 * 3) + 2]);
}

__global__ void March(float* devDensityCutoff,
	int* devVoxelArraySize,
	float* devNodeDensityArray,
	float* devNodePositionArray,
	int* devEdgeTable,
	int* devTriTable,
	float* devAllVertices,
	int* devVertCounter)
{
	int voxelIter = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(voxelIter < *devVoxelArraySize)
	{
		unsigned char voxelByte = 0x00;

		if (devNodeDensityArray[(voxelIter * 8) + 1] < *devDensityCutoff) voxelByte |= 1;
		if (devNodeDensityArray[(voxelIter * 8) + 5] < *devDensityCutoff) voxelByte |= 2;
		if (devNodeDensityArray[(voxelIter * 8) + 4] < *devDensityCutoff) voxelByte |= 4;
		if (devNodeDensityArray[(voxelIter * 8) + 0] < *devDensityCutoff) voxelByte |= 8;
		if (devNodeDensityArray[(voxelIter * 8) + 3] < *devDensityCutoff) voxelByte |= 16;
		if (devNodeDensityArray[(voxelIter * 8) + 7] < *devDensityCutoff) voxelByte |= 32;
		if (devNodeDensityArray[(voxelIter * 8) + 6] < *devDensityCutoff) voxelByte |= 64;
		if (devNodeDensityArray[(voxelIter * 8) + 2] < *devDensityCutoff) voxelByte |= 128;

		if (devEdgeTable[voxelByte] != 0)
		{
			float vertexArray[12];

			if (devEdgeTable[voxelByte] & 1)	//	AND operator
				AddInterpolatedVertex(voxelIter, 0, vertexArray, devNodePositionArray, devNodeDensityArray, 1, 5, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 2)
				AddInterpolatedVertex(voxelIter, 1, vertexArray, devNodePositionArray, devNodeDensityArray, 5, 4, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 4)
				AddInterpolatedVertex(voxelIter, 2, vertexArray, devNodePositionArray, devNodeDensityArray, 4, 0, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 8)
				AddInterpolatedVertex(voxelIter, 3, vertexArray, devNodePositionArray, devNodeDensityArray, 0, 1, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 16)
				AddInterpolatedVertex(voxelIter, 4, vertexArray, devNodePositionArray, devNodeDensityArray, 3, 7, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 32)
				AddInterpolatedVertex(voxelIter, 5, vertexArray, devNodePositionArray, devNodeDensityArray, 7, 6, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 64)
				AddInterpolatedVertex(voxelIter, 6, vertexArray, devNodePositionArray, devNodeDensityArray, 6, 2, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 128)
				AddInterpolatedVertex(voxelIter, 7, vertexArray, devNodePositionArray, devNodeDensityArray, 2, 3, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 256)
				AddInterpolatedVertex(voxelIter, 8, vertexArray, devNodePositionArray, devNodeDensityArray, 1, 3, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 512)
				AddInterpolatedVertex(voxelIter, 9, vertexArray, devNodePositionArray, devNodeDensityArray, 5, 7, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 1024)
				AddInterpolatedVertex(voxelIter, 10, vertexArray, devNodePositionArray, devNodeDensityArray, 4, 6, *devDensityCutoff);
			if (devEdgeTable[voxelByte] & 2048)
				AddInterpolatedVertex(voxelIter, 11, vertexArray, devNodePositionArray, devNodeDensityArray, 0, 2, *devDensityCutoff);
		
			for (int vertIter = 0; devTriTable[(voxelByte * 16) + vertIter] != -1; vertIter += 3)
			{
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 0) * 3) + 0] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter)] * 3) + 0];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 0) * 3) + 1] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter)] * 3) + 1];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 0) * 3) + 2] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter)] * 3) + 2];

				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 1) * 3) + 0] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 1)] * 3) + 0];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 1) * 3) + 1] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 1)] * 3) + 1];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 1) * 3) + 2] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 1)] * 3) + 2];

				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 2) * 3) + 0] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 2)] * 3) + 0];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 2) * 3) + 1] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 2)] * 3) + 1];
				devAllVertices[(voxelIter * (3 * 15)) + ((vertIter + 2) * 3) + 2] = vertexArray[(devTriTable[(voxelByte * 16) + (vertIter + 2)] * 3) + 2];

				atomicAdd(&devVertCounter[0], 3);
			}
		}
	}
}

struct is_non_zero
{
	__host__ __device__ bool operator()(const float x)
	{
		return x != 0;
	}
};

void RunMarchingCubes(float densityCutoff,
	Voxel* voxelArray,
	int voxelArraySize,
	XMFLOAT3* allVertices,
	int* allIndices,
	int& vertCounter,
	int* triTable,
	int* edgeTable,
	double& nodeParseTime,
	double& mallocTime,
	double& memcpyTime,
	double& marchTime,
	double& compactTime,
	double& freeTime)
{
	the_clock::time_point p1 = the_clock::now();

	//
	//	Create and Load intermediate arrays
	float* nodeDensityArray = new float[8 * voxelArraySize];
	float* nodePositionArray = new float[3 * 8 * voxelArraySize];
	
	for (int i = 0; i < voxelArraySize; ++i)
	{
		for (int j = 0; j < 8; ++j)
		{
			nodeDensityArray[(i * 8) + j] = voxelArray[i].getNode(j)->density;

			nodePositionArray[(i * (8 * 3)) + (j * 3) + 0] = voxelArray[i].getNode(j)->position.x;
			nodePositionArray[(i * (8 * 3)) + (j * 3) + 1] = voxelArray[i].getNode(j)->position.y;
			nodePositionArray[(i * (8 * 3)) + (j * 3) + 2] = voxelArray[i].getNode(j)->position.z;
		}
	}

	the_clock::time_point p2 = the_clock::now();
	nodeParseTime = duration_cast<microseconds>(p2 - p1).count();

	float* devDensityCutoff = 0;
	int* devVoxelArraySize = 0;
	float* devNodeDensityArray = 0;
	float* devNodePositionArray = 0;
	int* devEdgeTable = 0;
	int* devTriTable = 0;
	float* devAllVertices = 0;
	int* devVertCounter = 0;


	hipError_t cudaStatus;

	//
	//	Malloc
	cudaStatus = hipMallocManaged((void**)&devDensityCutoff, sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devVoxelArraySize, sizeof(int));
	if (cudaStatus != cudaStatus)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devNodeDensityArray, 8 * voxelArraySize * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devNodePositionArray, 3 * 8 * voxelArraySize * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devEdgeTable, 256 * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devTriTable, 256 * 16 * sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devVertCounter, sizeof(int));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMallocManaged((void**)&devAllVertices, 15 * 3 * voxelArraySize * sizeof(float));
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	the_clock::time_point p3 = the_clock::now();
	mallocTime = duration_cast<microseconds>(p3 - p2).count();

	//
	//	Initialise arrays with values
	cudaStatus = hipMemcpy(devDensityCutoff, &densityCutoff, sizeof(float), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	cudaStatus = hipMemcpy(devVoxelArraySize, &voxelArraySize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	cudaStatus = hipMemcpy(devNodeDensityArray, nodeDensityArray, 8 * voxelArraySize * sizeof(float), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	cudaStatus = hipMemcpy(devNodePositionArray, nodePositionArray, 3 * 8 * voxelArraySize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	cudaStatus = hipMemcpy(devEdgeTable, edgeTable, 256 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	cudaStatus = hipMemcpy(devTriTable, triTable, 256 * 16 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	//
	//	Delete intermediate dynamic arrays
	delete nodeDensityArray;
	nodeDensityArray = 0;

	delete nodePositionArray;
	nodePositionArray = 0;

	the_clock::time_point p4 = the_clock::now();
	memcpyTime = duration_cast<microseconds>(p4 - p3).count();

	//
	//	Optimise thread hierarchies
	int numThreads = voxelArraySize % 32 == 0 ? voxelArraySize : ((voxelArraySize / 32) + 1.0f) * 32;
	int numBlocks = 1;

	if (numThreads > 1024)
	{
		numBlocks = numThreads % 1024 == 0 ? (numThreads / 1024) : (numThreads / 1024) + 1;
		numThreads = numThreads / numBlocks;
	}

	dim3 blocks(numBlocks);
	dim3 threads(numThreads);

	//
	//	Run
	March << <blocks, threads>> > (devDensityCutoff,
		devVoxelArraySize,
		devNodeDensityArray,
		devNodePositionArray,
		devEdgeTable,
		devTriTable,
		devAllVertices,
		devVertCounter);

	//
	//	Check error
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

	//
	//	Sync
	hipDeviceSynchronize();

	the_clock::time_point p5 = the_clock::now();
	marchTime = duration_cast<microseconds>(p5 - p4).count();

	//
	//	Compact verts and indices
	if (cudaStatus == hipSuccess)
	{

		thrust::device_vector<float> t_devAllVertices(devAllVertices, devAllVertices + (voxelArraySize * 3 * 15));

		thrust::device_vector<float> t_compactAllVertices(voxelArraySize * 3 * 15, 0);

		thrust::copy_if(thrust::device, t_devAllVertices.begin(), t_devAllVertices.end(), t_compactAllVertices.begin(),is_non_zero());

		thrust::copy(t_compactAllVertices.begin(), t_compactAllVertices.end(), devAllVertices);

		cudaStatus = hipMemcpy(allVertices, devAllVertices, 15 * 3 * voxelArraySize * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "hipMemcpy failed!");

		cudaStatus = hipMemcpy(&vertCounter, devVertCounter,sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "hipMemcpy failed!");
		
		thrust::sequence(thrust::host, allIndices, allIndices + vertCounter, 0);
	}

	the_clock::time_point p6 = the_clock::now();
	compactTime = duration_cast<microseconds>(p6 - p5).count();

	//
	//	Free
	hipFree(devDensityCutoff);
	hipFree(devVoxelArraySize);
	hipFree(devNodeDensityArray);
	hipFree(devNodePositionArray);
	hipFree(devEdgeTable);
	hipFree(devTriTable);
	hipFree(devAllVertices);
	hipFree(devVertCounter);

	the_clock::time_point p7 = the_clock::now();
	freeTime = duration_cast<microseconds>(p7 - p6).count();
}

#endif